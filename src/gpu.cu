#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;

__global__ void BellmanFord(int *B, int *C, int *rowIndices, int* columnIndices, int *updatedVertices, int edgeCount) {
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = threadID; i < edgeCount; i += stride) {
        int source = rowIndices[i];
        int target = columnIndices[i];

        if (B[source] + 1 < C[target]) {
            atomicMin(&C[target], B[source] + 1);
            updatedVertices[target] = 1;
        }
    }
}

void runGPU(vector<int> &B, vector<int> &C, vector<int> &rowIndices, vector<int> &columnIndices, vector<int> &updatedVertices) {

    // arbitrary blocksize
    int edgeCount = rowIndices.size();
    int blockSize = 256;
<<<<<<< HEAD
    int numBlocks = (numEdges + blockSize - 1) / blockSize;

    int *d_rows, *d_columns, *d_updates, *d_dists, *d_sources;

    // intialize device array pointers
    hipMalloc((void **) &d_rows, rowIndices.size() * sizeof(int));
    hipMalloc((void **) &d_columns, columnIndices.size() * sizeof(int));
    hipMalloc((void **) &d_dists, B.size() * sizeof(int));
    hipMalloc((void **) &d_sources, C.size() * sizeof(int));
    hipMalloc((void **) &d_updates, updates.size() * sizeof(int));

    // load data into device 
    hipMemcpy(d_rows, rowIndices.data(), rowIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_columns, columnIndices.data(), columnIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dists, B.data(), B.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sources, C.data(), C.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_updates, updates.data(), updates.size() * sizeof(int), hipMemcpyHostToDevice);

    // begin iteration
    BellmanFord<<<numBlocks, blockSize>>>(numVertices, numEdges, d_rows, d_columns, d_dists, d_sources, d_updates);

    // update host memory
    hipMemcpy(&B, d_dists, B.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&C, d_sources, C.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&updates, d_updates, updates.size() * sizeof(int), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_rows);
    hipFree(d_columns);
    hipFree(d_dists);
    hipFree(d_sources);
    hipFree(d_updates);
=======
    int numBlocks = (edgeCount + blockSize - 1) / blockSize;

    int *deviceB, *deviceC, *deviceRowIndices, *deviceColumnIndices, *deviceUpdatedVertices;

    hipMalloc((void **) &deviceB, B.size() * sizeof(int));
    hipMalloc((void **) &deviceC, C.size() * sizeof(int));
    hipMalloc((void **) &deviceRowIndices, rowIndices.size() * sizeof(int));
    hipMalloc((void **) &deviceColumnIndices, columnIndices.size() * sizeof(int));
    hipMalloc((void **) &deviceUpdatedVertices, updatedVertices.size() * sizeof(int));

    hipMemcpy(deviceB, B.data(), B.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceC, C.data(), C.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceRowIndices, rowIndices.data(), rowIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceColumnIndices, columnIndices.data(), columnIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceUpdatedVertices, updatedVertices.data(), updatedVertices.size() * sizeof(int), hipMemcpyHostToDevice);
    
    BellmanFord<<<numBlocks, blockSize>>>(deviceB, deviceC, deviceRowIndices, deviceColumnIndices, deviceUpdatedVertices, edgeCount);

    hipMemcpy(B.data(), deviceB, B.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(C.data(), deviceC, C.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(updatedVertices.data(), deviceUpdatedVertices, updatedVertices.size() * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceB);
    hipFree(deviceC);
    hipFree(deviceRowIndices);
    hipFree(deviceColumnIndices);
    hipFree(deviceUpdatedVertices);
>>>>>>> 4c8a9273255ed9e18c3233d48288fded42fdda16
}
