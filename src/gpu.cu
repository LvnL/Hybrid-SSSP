#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;

__global__ void BellmanFord(int numVertices, int numEdges, int *rows, int *columns, float *vals, float *dists, float *sources) {
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // int update = FALSE;

    for (int i=t_id; i<numEdges; i+=stride) {
        int u = rows[i];
        int v = columns[i];
        float val = vals[i];

        if (dists[v] + val < sources[u]) {
            sources[u] = dists[v] + val;
            // update = TRUE; // non-functional for now
        }
    }

    // swap sources and dists here or in main loop
    float *tmp = dists;
    dists = sources;
    sources = tmp;

    /*
    long threadID = (long) tn;
	int blockSize = numberOfRows / numberOfThreads;

	for (int source = threadID * blockSize; source < (threadID + 1) * blockSize; source++) {
		for (int j = rowIndices[source]; j < (source == numberOfRows - 1 ? columnIndices.size() : rowIndices[source + 1]); j++) {
			int target = columnIndices[j];
			float value = values[j];
			if (B[source] + value < C[target]) {
				C[target] = B[source] + value;
				updated = true;
			}
		}
	}
    */
}

void runGPU(vector<float> &B, vector<float> &C, vector<float> &values, vector<int> &rowIndices, vector<int> &columnIndices, int numVertices) {

    cout << "Begin GPU runtime..." << endl;

    // arbitrary blocksize
    int numEdges = rowIndices.size();
    int blockSize = 256;
    int numBlocks = (numEdges + blockSize - 1) / blockSize;

    int *d_rows, *d_columns;
    float *d_vals, *d_dists, *d_sources;

    // intialize device array pointers
    hipMalloc((void **) &d_rows, rowIndices.size() * sizeof(int));
    hipMalloc((void **) &d_columns, columnIndices.size() * sizeof(int));
    hipMalloc((void **) &d_vals, values.size() * sizeof(float));
    hipMalloc((void **) &d_dists, B.size() * sizeof(float));
    hipMalloc((void **) &d_sources, C.size() * sizeof(float));

    // load data into device 
    hipMemcpy(d_rows, rowIndices.data(), rowIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_columns, columnIndices.data(), columnIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vals, values.data(), values.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dists, B.data(), B.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sources, C.data(), C.size() * sizeof(float), hipMemcpyHostToDevice);

    // begin iteration
    BellmanFord<<<numBlocks, blockSize>>>(numVertices, numEdges, d_rows, d_columns, d_vals, d_dists, d_sources);

    // update host memory
    hipMemcpy(&B, d_dists, B.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&C, d_sources, C.size() * sizeof(float), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_rows);
    hipFree(d_columns);
    hipFree(d_vals);
    hipFree(d_dists);
    hipFree(d_sources);
}
