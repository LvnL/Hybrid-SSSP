#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;

__global__ void BellmanFord(int numVertices, int numEdges, int *rows, int *columns, float *dists, float *sources, int *updates) {
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i=t_id; i<numEdges; i+=stride) {
        int u = rows[i];
        int v = columns[i];

        if (dists[v] + 1 < sources[u]) {
            atomicMin(sources[u], dists[v] + 1);
            atomicMax(updates[u], 1);
            // sources[u] = dists[v] + 1;
            // updates[u] = 1;
        }
    }
}

void runGPU(vector<float> &B, vector<float> &C, vector<int> &rowIndices, vector<int> &columnIndices, int numVertices, vector<int> &updates) {

    // arbitrary blocksize
    int numEdges = rowIndices.size();
    int blockSize = 256;
    int numBlocks = (numEdges + blockSize - 1) / blockSize;

    int *d_rows, *d_columns, *d_updates;
    float *d_dists, *d_sources;

    // intialize device array pointers
    hipMalloc((void **) &d_rows, rowIndices.size() * sizeof(int));
    hipMalloc((void **) &d_columns, columnIndices.size() * sizeof(int));
    hipMalloc((void **) &d_dists, B.size() * sizeof(float));
    hipMalloc((void **) &d_sources, C.size() * sizeof(float));
    hipMalloc((void **) &d_updates, updates.size() * sizeof(int));

    // load data into device 
    hipMemcpy(d_rows, rowIndices.data(), rowIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_columns, columnIndices.data(), columnIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dists, B.data(), B.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sources, C.data(), C.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_updates, updates.data(), updates.size() * sizeof(int), hipMemcpyHostToDevice);

    // begin iteration
    BellmanFord<<<numBlocks, blockSize>>>(numVertices, numEdges, d_rows, d_columns, d_dists, d_sources, d_updates);

    // update host memory
    hipMemcpy(&B, d_dists, B.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&C, d_sources, C.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&updates, d_updates, updates.size() * sizeof(int), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_rows);
    hipFree(d_columns);
    hipFree(d_dists);
    hipFree(d_sources);
    hipFree(d_updates);
}
