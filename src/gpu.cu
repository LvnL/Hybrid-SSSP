#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;

__global__ void BellmanFord(int *B, int *C, int *rows, int* columns, int *updatedVertices, int edgeCount) {
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = threadID; i < edgeCount; i += stride) {
        int source = rows[i];
        int target = columns[i];

        if (B[source] + 1 < C[target]) {
            atomicMin(&C[target], B[source] + 1);
            updatedVertices[target] = 1;
        }
    }
}

void runGPU(vector<int> &B, vector<int> &C, vector<int> &rows, vector<int> &columns, vector<int> &updatedVertices) {
    // arbitrary blocksize
    int edgeCount = rows.size();
    int blockSize = 256;
<<<<<<< HEAD
    int numBlocks = (numEdges + blockSize - 1) / blockSize;

    int *d_rows, *d_columns, *d_updates, *d_dists, *d_sources;

    // intialize device array pointers
    hipMalloc((void **) &d_rows, rowIndices.size() * sizeof(int));
    hipMalloc((void **) &d_columns, columnIndices.size() * sizeof(int));
    hipMalloc((void **) &d_dists, B.size() * sizeof(int));
    hipMalloc((void **) &d_sources, C.size() * sizeof(int));
    hipMalloc((void **) &d_updates, updates.size() * sizeof(int));

    // load data into device 
    hipMemcpy(d_rows, rowIndices.data(), rowIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_columns, columnIndices.data(), columnIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dists, B.data(), B.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_sources, C.data(), C.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_updates, updates.data(), updates.size() * sizeof(int), hipMemcpyHostToDevice);

    // begin iteration
    BellmanFord<<<numBlocks, blockSize>>>(numVertices, numEdges, d_rows, d_columns, d_dists, d_sources, d_updates);

    // update host memory
    hipMemcpy(&B, d_dists, B.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&C, d_sources, C.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&updates, d_updates, updates.size() * sizeof(int), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_rows);
    hipFree(d_columns);
    hipFree(d_dists);
    hipFree(d_sources);
    hipFree(d_updates);
=======
    int numBlocks = (edgeCount + blockSize - 1) / blockSize;

    int *deviceB, *deviceC, *deviceRows, *deviceColumns, *deviceUpdatedVertices;

    hipMalloc((void **) &deviceB, B.size() * sizeof(int));
    hipMalloc((void **) &deviceC, C.size() * sizeof(int));
    hipMalloc((void **) &deviceRows, rows.size() * sizeof(int));
    hipMalloc((void **) &deviceColumns, columns.size() * sizeof(int));
    hipMalloc((void **) &deviceUpdatedVertices, updatedVertices.size() * sizeof(int));

    hipMemcpy(deviceB, B.data(), B.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceC, C.data(), C.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceRows, rows.data(), rows.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceColumns, columns.data(), columns.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceUpdatedVertices, updatedVertices.data(), updatedVertices.size() * sizeof(int), hipMemcpyHostToDevice);
    
    BellmanFord<<<numBlocks, blockSize>>>(deviceB, deviceC, deviceRows, deviceColumns, deviceUpdatedVertices, edgeCount);

    hipMemcpy(B.data(), deviceB, B.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(C.data(), deviceC, C.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(rows.data(), deviceRows, rows.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(columns.data(), deviceColumns, columns.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(updatedVertices.data(), deviceUpdatedVertices, updatedVertices.size() * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceB);
    hipFree(deviceC);
    hipFree(deviceRows);
    hipFree(deviceColumns);
    hipFree(deviceUpdatedVertices);
>>>>>>> 4c8a9273255ed9e18c3233d48288fded42fdda16
}
