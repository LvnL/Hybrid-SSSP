#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;

__global__ void BellmanFord(int *B, int *C, int *rowIndices, int* columnIndices, int *updatedVertices, int edgeCount) {
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = threadID; i < edgeCount; i += stride) {
        int source = rowIndices[i];
        int target = columnIndices[i];

        if (B[source] + 1 < C[target]) {
            atomicMin(&C[target], B[source] + 1);
            updatedVertices[target] = 1;
        }
    }
}

void runGPU(vector<int> &B, vector<int> &C, vector<int> &rowIndices, vector<int> &columnIndices, vector<int> &updatedVertices) {

    // arbitrary blocksize
    int edgeCount = rowIndices.size();
    int blockSize = 256;
    int numBlocks = (edgeCount + blockSize - 1) / blockSize;

    int *deviceB, *deviceC, *deviceRowIndices, *deviceColumnIndices, *deviceUpdatedVertices;

    hipMalloc((void **) &deviceB, B.size() * sizeof(int));
    hipMalloc((void **) &deviceC, C.size() * sizeof(int));
    hipMalloc((void **) &deviceRowIndices, rowIndices.size() * sizeof(int));
    hipMalloc((void **) &deviceColumnIndices, columnIndices.size() * sizeof(int));
    hipMalloc((void **) &deviceUpdatedVertices, updatedVertices.size() * sizeof(int));

    hipMemcpy(deviceB, B.data(), B.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceC, C.data(), C.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceRowIndices, rowIndices.data(), rowIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceColumnIndices, columnIndices.data(), columnIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceUpdatedVertices, updatedVertices.data(), updatedVertices.size() * sizeof(int), hipMemcpyHostToDevice);
    
    BellmanFord<<<numBlocks, blockSize>>>(deviceB, deviceC, deviceRowIndices, deviceColumnIndices, deviceUpdatedVertices, edgeCount);

    hipMemcpy(B.data(), deviceB, B.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(C.data(), deviceC, C.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(updatedVertices.data(), deviceUpdatedVertices, updatedVertices.size() * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceB);
    hipFree(deviceC);
    hipFree(deviceRowIndices);
    hipFree(deviceColumnIndices);
    hipFree(deviceUpdatedVertices);
}
