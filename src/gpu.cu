#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;

__global__ void BellmanFord(int *B, int *C, int *rows, int* columns, int *updatedVertices, int vertexCount) {
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = threadID; i < vertexCount; i += stride) {
        int source = rows[i];
        int target = columns[i];

        if (B[source] + 1 < C[target]) {
            atomicMin(&C[target], B[source] + 1);
            updatedVertices[target] = 1;
        }
    }
}

void runGPU(vector<int> &B, vector<int> &C, vector<int> &rows, vector<int> &columns, vector<int> &updatedVertices) {
    // arbitrary blocksize
    int blockSize = 256;
    int numBlocks = (rows.size() + blockSize - 1) / blockSize;

    int *deviceB, *deviceC, *deviceRows, *deviceColumns, *deviceUpdatedVertices;

    // initialize device array pointers
    hipMalloc((void **) &deviceB, B.size() * sizeof(int));
    hipMalloc((void **) &deviceC, C.size() * sizeof(int));
    hipMalloc((void **) &deviceRows, rows.size() * sizeof(int));
    hipMalloc((void **) &deviceColumns, columns.size() * sizeof(int));
    hipMalloc((void **) &deviceUpdatedVertices, updatedVertices.size() * sizeof(int));

    // load data into device
    hipMemcpy(deviceB, B.data(), B.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceC, C.data(), C.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceRows, rows.data(), rows.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceColumns, columns.data(), columns.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceUpdatedVertices, updatedVertices.data(), updatedVertices.size() * sizeof(int), hipMemcpyHostToDevice);
    
    // begin iteration via kernel
    BellmanFord<<<numBlocks, blockSize>>>(deviceB, deviceC, deviceRows, deviceColumns, deviceUpdatedVertices, rows.size());

    // update host memory
    hipMemcpy(B.data(), deviceB, B.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(C.data(), deviceC, C.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(rows.data(), deviceRows, rows.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(columns.data(), deviceColumns, columns.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(updatedVertices.data(), deviceUpdatedVertices, updatedVertices.size() * sizeof(int), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(deviceB);
    hipFree(deviceC);
    hipFree(deviceRows);
    hipFree(deviceColumns);
    hipFree(deviceUpdatedVertices);
}
