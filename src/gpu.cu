#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;

__global__ void BellmanFord(int numVertices, int numEdges, int *rows, int *columns, float *vals, float *dists, float *sources, int *updates) {
    int t_id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i=t_id; i<numEdges; i+=stride) {
        int u = rows[i];
        int v = columns[i];
        float val = vals[i];

        if (dists[v] + val < sources[u]) {
            sources[u] = dists[v] + val;
            updates[u] = 1;
        }
    }
}

void runGPU(vector<float> &B, vector<float> &C, vector<float> &values, vector<int> &rowIndices, vector<int> &columnIndices, int numVertices, vector<int> &updates) {

    // arbitrary blocksize
    int numEdges = rowIndices.size();
    int blockSize = 256;
    int numBlocks = (numEdges + blockSize - 1) / blockSize;

    int *d_rows, *d_columns, *d_updates;
    float *d_vals, *d_dists, *d_sources;

    // intialize device array pointers
    hipMalloc((void **) &d_rows, rowIndices.size() * sizeof(int));
    hipMalloc((void **) &d_columns, columnIndices.size() * sizeof(int));
    hipMalloc((void **) &d_vals, values.size() * sizeof(float));
    hipMalloc((void **) &d_dists, B.size() * sizeof(float));
    hipMalloc((void **) &d_sources, C.size() * sizeof(float));
    hipMalloc((void **) &d_updates, updates.size() * sizeof(int));

    // load data into device 
    hipMemcpy(d_rows, rowIndices.data(), rowIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_columns, columnIndices.data(), columnIndices.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_vals, values.data(), values.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dists, B.data(), B.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sources, C.data(), C.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_updates, updates.data(), updates.size() * sizeof(int), hipMemcpyHostToDevice);

    // begin iteration
    BellmanFord<<<numBlocks, blockSize>>>(numVertices, numEdges, d_rows, d_columns, d_vals, d_dists, d_sources, d_updates);

    // update host memory
    hipMemcpy(&B, d_dists, B.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&C, d_sources, C.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&updates, d_updates, updates.size() * sizeof(int), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_rows);
    hipFree(d_columns);
    hipFree(d_vals);
    hipFree(d_dists);
    hipFree(d_sources);
    hipFree(d_updates);
}
