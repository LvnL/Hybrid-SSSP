#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>

using namespace std;

__global__ void BellmanFord(int *B, int *C, int *rows, int* columns, int *updatedVertices, int edgeCount) {
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = threadID; i < edgeCount; i += stride) {
        int source = rows[i];
        int target = columns[i];

        if (B[source] + 1 < C[target]) {
            atomicMin(&C[target], B[source] + 1);
            updatedVertices[target] = 1;
        }
    }
}

void runGPU(vector<int> &B, vector<int> &C, vector<int> &rows, vector<int> &columns, vector<int> &updatedVertices) {
    // arbitrary blocksize
    int edgeCount = rows.size();
    int blockSize = 256;
    int numBlocks = (edgeCount + blockSize - 1) / blockSize;

    int *deviceB, *deviceC, *deviceRows, *deviceColumns, *deviceUpdatedVertices;

    hipMalloc((void **) &deviceB, B.size() * sizeof(int));
    hipMalloc((void **) &deviceC, C.size() * sizeof(int));
    hipMalloc((void **) &deviceRows, rows.size() * sizeof(int));
    hipMalloc((void **) &deviceColumns, columns.size() * sizeof(int));
    hipMalloc((void **) &deviceUpdatedVertices, updatedVertices.size() * sizeof(int));

    hipMemcpy(deviceB, B.data(), B.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceC, C.data(), C.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceRows, rows.data(), rows.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceColumns, columns.data(), columns.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceUpdatedVertices, updatedVertices.data(), updatedVertices.size() * sizeof(int), hipMemcpyHostToDevice);
    
    BellmanFord<<<numBlocks, blockSize>>>(deviceB, deviceC, deviceRows, deviceColumns, deviceUpdatedVertices, edgeCount);

    hipMemcpy(B.data(), deviceB, B.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(C.data(), deviceC, C.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(rows.data(), deviceRows, rows.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(columns.data(), deviceColumns, columns.size() * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(updatedVertices.data(), deviceUpdatedVertices, updatedVertices.size() * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceB);
    hipFree(deviceC);
    hipFree(deviceRows);
    hipFree(deviceColumns);
    hipFree(deviceUpdatedVertices);
}
